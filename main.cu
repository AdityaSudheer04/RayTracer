#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hittable_list.h"
#include "camera.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ float hit_sphere(const vec3& center, float radius, const ray& r) {
    vec3 oc = r.origin() - center;
    float a = dot(r.direction(), r.direction());
    float b = 2.0f * dot(r.direction(), oc);
    float c = dot(oc, oc) - radius * radius;
    float discriminant = b * b - 4.0f * a * c;
    if (discriminant < 0) return -1.0f;
    else return ( ( - b - sqrt(discriminant)) / (2.0f * a) );
}

__device__ vec3 color(const ray& r, hittable **d_world) {
    hit_record rec;
    if ((*d_world)->hit(r, 0.0, FLT_MAX, rec)) {
        return 0.5f * vec3(rec.normal.x() + 1.0f, rec.normal.y() + 1.0f, rec.normal.z() + 1.0f);
    }
    vec3 unit_direction = unit_vector(r.direction());
    float a = 0.5f * (unit_direction.y() + 1.0f);   //Writing f required as doube precision is default
    return (1.0f - a) * vec3(1.0, 1.0, 1.0) + a * vec3(0.3, 0.3, 1.0);
}   

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3* fb, int max_x, int max_y, int samples_per_pix,
                       camera** cam, hittable **d_world, hiprandState* rand_state) {
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0, 0, 0);
    for (int s = 0; s < samples_per_pix; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v);
        col += color(r, d_world);
    }
    fb[pixel_index] = col / float(samples_per_pix);
}

__global__ void create_world(hittable** d_list, hittable** d_world, camera** d_camera) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_list) = new sphere(vec3(0, 0, -2), 0.5);
        *(d_list + 1) = new sphere(vec3(0, -100.5, -2), 100);
        *(d_list + 2) = new sphere(vec3(-2, 0, -3), 0.5);
        *(d_world) = new hittable_list(d_list, 3);
        *d_camera = new camera();
    }
}

__global__ void free_world(hittable** d_list, hittable** d_world, camera** d_camera) {
    delete* (d_list);
    delete* (d_list + 1);
    delete* (d_list + 2);
    delete* d_world;
    delete* d_camera;
}

int main() {
    int image_width = 1200;
    int image_height = 600;
    int samples_per_pix = 100;
    int tx = 16;
    int ty = 16;

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width * image_height;
    size_t fb_size = num_pixels * sizeof(vec3);

    // allocate Frame Buffer
    // Frame Buffer is a vec3 array that contains all the values of RGB fo each pixel
    // from top left to bottom right
    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // allocate random state
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));

    // make our world of hittables and camera
    hittable** d_list;  // d prefix for device only
    checkCudaErrors(hipMalloc((void**)&d_list, 3 * sizeof(hittable*)));

    hittable** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hittable*)));

    camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));

    create_world << <1, 1 >> > (d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    
    clock_t start, stop;
    start = clock();

    // Render our buffer
    dim3 blocks(image_width / tx + 1, image_height / ty + 1);
    dim3 threads(tx, ty);
    render << <blocks, threads >> > (fb, image_width, image_height, 
                                    samples_per_pix,
                                    d_camera,
                                    d_world,
                                    d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::cout << "P3\n" << image_width << " " << image_height << "\n255\n";
    for (int j = image_height - 1; j >= 0; j--) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * image_width + i;
            int ir = int(255.99 * fb[pixel_index].x());
            int ig = int(255.99 * fb[pixel_index].y());
            int ib = int(255.99 * fb[pixel_index].z());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world << <1, 1 >> > (d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();
}