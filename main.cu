#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hittable_list.h"
#include "camera.h"
#include "material.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}


// Turning the followin recursive code to itereative with a limit of 50
__device__ vec3 color(const ray& r, hittable **d_world, hiprandState* local_rand_state) {
    ray curr_ray = r;
    vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);

    for (int i{ 1 }; i <= 50; i++) {
        hit_record rec;
        // Diffusion with attenuation 0.5
        // shadow acne removal by making t_min as 0.001
        if ((*d_world)->hit(curr_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if (rec.mat_ptr->scatter(curr_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation = cur_attenuation * attenuation;
                curr_ray = scattered;
            }
            else {
                return vec3(0.0, 0.0, 0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(curr_ray.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            vec3 c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.6, 1.0);
            return cur_attenuation * c;
        }
    }

    return vec3(0.0, 0.0, 0.0); //exceeeds recursion limit
}   

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3* fb, int max_x, int max_y, int samples_per_pix,
                       camera** cam, hittable **d_world, hiprandState* rand_state) {
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0, 0, 0);
    for (int s = 0; s < samples_per_pix; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v);
        col += color(r, d_world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(samples_per_pix);

    // Gamma correction
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);

    fb[pixel_index] = col;
}

__global__ void create_world(hittable** d_list, hittable** d_world, camera** d_camera) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_list[0] = new sphere(vec3(0, 0, -1), 0.5,
                               new lambertian(vec3(0.8, 0.3, 0.3)));
        d_list[1] = new sphere(vec3(0, -100.5, -1), 100,
                               new lambertian(vec3(0.8, 0.8, 0.0)));
        d_list[2] = new sphere(vec3(1, 0, -1), 0.5,
                               new metal(vec3(0.8, 0.6, 0.2), 1.0));
        d_list[3] = new sphere(vec3(-1, 0, -1), 0.5,
                               new metal(vec3(0.8, 0.8, 0.8), 0.3));
        *(d_world) = new hittable_list(d_list, 4);
        *d_camera = new camera();
    }
}

__global__ void free_world(hittable** d_list, hittable** d_world, camera** d_camera) {
    for (int i{ 0 }; i < 4; i++) {
        delete ((sphere*)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete* d_world;
    delete* d_camera;
}

int main() {
    int image_width = 1200;
    int image_height = 600;
    int samples_per_pix = 100;
    int tx = 16;
    int ty = 16;

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width * image_height;
    size_t fb_size = num_pixels * sizeof(vec3);

    // allocate Frame Buffer
    // Frame Buffer is a vec3 array that contains all the values of RGB fo each pixel
    // from top left to bottom right
    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // allocate random state
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));

    // make our world of hittables and camera
    hittable** d_list;  // d prefix for device only
    checkCudaErrors(hipMalloc((void**)&d_list, 4 * sizeof(hittable*)));

    hittable** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hittable*)));

    camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));

    create_world << <1, 1 >> > (d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    
    clock_t start, stop;
    start = clock();

    // Render our buffer
    dim3 blocks(image_width / tx + 1, image_height / ty + 1);
    dim3 threads(tx, ty);
    render_init << <blocks, threads >> > (image_width, image_height, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render << <blocks, threads >> > (fb, image_width, image_height, 
                                    samples_per_pix,
                                    d_camera,
                                    d_world,
                                    d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::cout << "P3\n" << image_width << " " << image_height << "\n255\n";
    for (int j = image_height - 1; j >= 0; j--) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * image_width + i;
            int ir = int(255.99 * fb[pixel_index].x());
            int ig = int(255.99 * fb[pixel_index].y());
            int ib = int(255.99 * fb[pixel_index].z());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world << <1, 1 >> > (d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();
}