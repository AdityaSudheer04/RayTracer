#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include "vec3.h"
#include "ray.h"
// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ bool hit_sphere(const vec3& center, float radius, const ray& r) {
    vec3 oc = r.origin() - center;
    float a = dot(r.direction(), r.direction());
    float b = 2.0f * dot(r.direction(), oc);
    float c = dot(oc, oc) - radius * radius;
    float discriminant = b * b - 4.0f * a * c;
    return (discriminant >= 0.0f);
}

__device__ vec3 color(const ray& r) {
    if (hit_sphere(vec3(0, 0, -1), 0.5, r)) {
        return vec3(0, 1, 1);
    }
    vec3 unit_direction = unit_vector(r.direction());
    float t = 0.5f * (unit_direction.y() + 1.0f);   //Writing f required as doube precision is default
    return (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.3, 0.3, 1.0);
}   

__global__ void render(vec3* fb, int max_x, int max_y, vec3 lower_left_corner, vec3 horizontal,
                        vec3 vertical, vec3 origin) {
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    float u = float(i) / max_x;
    float v = float(j) / max_y;
    
    ray r(origin, lower_left_corner + u * horizontal + v * vertical);
    fb[pixel_index] = color(r);
}

int main() {
    int image_width = 1200;
    int image_height = 600;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width * image_height;
    size_t fb_size = num_pixels * sizeof(vec3);

    // allocate Frame Buffer
    // Frame Buffer is a float array that contains all the values of RGB fo each pixel
    // from top left to bottom right
    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    clock_t start, stop;
    start = clock();

    // Render our buffer
    dim3 blocks(image_width / tx + 1, image_height / ty + 1);
    dim3 threads(tx, ty);
    render << <blocks, threads >> > (fb, image_width, image_height, 
                                    vec3(-2.0, -1.0, -1.0),
                                    vec3(4.0, 0.0, 0.0),
                                    vec3(0.0, 2.0, 0.0),
                                    vec3(0.0, 0.0, 0.0));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::cout << "P3\n" << image_width << " " << image_height << "\n255\n";
    for (int j = image_height - 1; j >= 0; j--) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * image_width + i;
            int ir = int(255.99 * fb[pixel_index].x());
            int ig = int(255.99 * fb[pixel_index].y());
            int ib = int(255.99 * fb[pixel_index].z());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    checkCudaErrors(hipFree(fb));
}