#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hittable_list.h"
#include "camera.h"
#include "material.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}


// Turning the followin recursive code to itereative with a limit of 50
__device__ vec3 color(const ray& r, hittable** d_world, hiprandState* local_rand_state) {
    ray curr_ray = r;
    vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);

    for (int i{ 1 }; i <= 50; i++) {
        hit_record rec;
        // Diffusion with attenuation 0.5
        // shadow acne removal by making t_min as 0.001
        if ((*d_world)->hit(curr_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if (rec.mat_ptr->scatter(curr_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation = cur_attenuation * attenuation;
                curr_ray = scattered;
            }
            else {
                return vec3(0.0, 0.0, 0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(curr_ray.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            vec3 c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.6, 1.0);
            return cur_attenuation * c;
        }
    }

    return vec3(0.0, 0.0, 0.0); //exceeeds recursion limit
}

__global__ void rand_init(hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3* fb, int max_x, int max_y, int samples_per_pix,
    camera** cam, hittable** d_world, hiprandState* rand_state) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0, 0, 0);
    for (int s = 0; s < samples_per_pix; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, d_world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(samples_per_pix);

    // Gamma correction
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);

    fb[pixel_index] = col;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hittable** d_list, hittable** d_world, camera** d_camera, int image_width,
    int image_height, hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new sphere(vec3(0, -10000.0, -1), 10000,
            new lambertian(vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for (int a = -11; a < 11; a++) {
            for (int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a + RND, 0.2, b + RND);
                if (choose_mat < 0.8f) {
                    d_list[i++] = new sphere(center, 0.2,
                        new lambertian(vec3(RND * RND, RND * RND, RND * RND)));
                }
                else if (choose_mat < 0.95f) {
                    d_list[i++] = new sphere(center, 0.2,
                        new metal(vec3(0.5f * (1.0f + RND), 0.5f * (1.0f + RND), 0.5f * (1.0f + RND)), 0.5f * RND));
                }
                else {
                    d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
        d_list[i++] = new sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world = new hittable_list(d_list, 22 * 22 + 1 + 3);

        vec3 lookfrom(3, 2, 12);
        vec3 lookat(0, 0, 0);
        float dist_to_focus = 10.0f;  (lookfrom - lookat).length();
        float aperture = 0.1f;
        *d_camera = new camera(lookfrom,
                                lookat,
                                vec3(0, 1, 0),
                                30.0,
                                float(image_width) / float(image_height),
                                aperture,
                                dist_to_focus);
    }
}

__global__ void free_world(hittable** d_list, hittable** d_world, camera** d_camera) {
    for (int i{ 0 }; i < 22 * 22 + 1 + 3; i++) {
        delete ((sphere*)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete* d_world;
    delete* d_camera;
}

int main() {
    int image_width = 1920;
    int image_height = 1080;
    int samples_per_pix = 10;
    int tx = 16;
    int ty = 16;

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width * image_height;
    size_t fb_size = num_pixels * sizeof(vec3);

    // allocate Frame Buffer
    // Frame Buffer is a vec3 array that contains all the values of RGB fo each pixel
    // from top left to bottom right
    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // allocate random state
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));

    hiprandState* d_rand_state2;
    checkCudaErrors(hipMalloc((void**)&d_rand_state2, 1 * sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init << <1, 1 >> > (d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hittables and camera
    hittable** d_list;  // d prefix for device only
    int num_hitables = 22 * 22 + 1 + 3;
    checkCudaErrors(hipMalloc((void**)&d_list, num_hitables * sizeof(hittable*)));

    hittable** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hittable*)));

    camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));

    create_world << <1, 1 >> > (d_list, d_world, d_camera, image_width, image_height, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());


    clock_t start, stop;
    start = clock();

    // Render our buffer
    dim3 blocks(image_width / tx + 1, image_height / ty + 1);
    dim3 threads(tx, ty);
    render_init << <blocks, threads >> > (image_width, image_height, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render << <blocks, threads >> > (fb, image_width, image_height,
        samples_per_pix,
        d_camera,
        d_world,
        d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::cout << "P3\n" << image_width << " " << image_height << "\n255\n";
    for (int j = image_height - 1; j >= 0; j--) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * image_width + i;
            int ir = int(255.99 * fb[pixel_index].x());
            int ig = int(255.99 * fb[pixel_index].y());
            int ib = int(255.99 * fb[pixel_index].z());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world << <1, 1 >> > (d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();
}